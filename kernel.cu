#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <cstring>

__global__ void sum_array_gpu(int* a, int* b, int* c, int size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

//host code, ran by CPU 
int main()
{

	int size = 10000;

	int block_size = 128;
	int NO_BYTES = size * sizeof(int);
	
	//host pointers
	int* h_a, * h_b, * gpu_results, *h_c;

	//allocate memory for host pointers
	h_a = (int*)malloc(NO_BYTES);
	h_b = (int*)malloc(NO_BYTES);
	gpu_results = (int*)malloc(NO_BYTES);
	h_c = (int*)malloc(NO_BYTES);

	//initialize host pointer
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		h_a[i] = (int)(rand() & 0xFF);
	}

	for (int i = 0; i < size; i++)
	{
		h_b[i] = (int)(rand() & 0xFF);
	}

	sum_array_cpu(h_a, h_b, h_c, size);

	memset(gpu_results, 0, NO_BYTES);

	//device pointer
	int* d_a, * d_b, * d_c;
	hipMalloc((int**)&d_a, NO_BYTES);
	hipMalloc((int**)&d_b, NO_BYTES);
	hipMalloc((int**)&d_c, NO_BYTES);

	//memory transfer from host to device
	hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

	//launching the grid
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1); //adding 1 gurantees that we gonna have more threads than array size

	sum_array_gpu <<< grid, block >> > (d_a, d_b, d_c, size);

	hipDeviceSynchronize();
	
	//memory transfer back to host
	hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);

	//array comparison
	{
		for (int i = 0; i < size; i++)
		{
			if (gpu_results[i] != h_c[i])
			{
				printf("Arrays are different\n");

			}
		}
		printf("Arrays are the same \n");
	}



	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_c);

	free(gpu_results);
	free(h_a);
	free(h_b);
	
	return 0;
}
